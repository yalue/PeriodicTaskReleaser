#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <sched.h>
#include <errno.h>

extern "C" {
#include "../gpusync.h"
}

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
    vectorAdd(const float *A, const float *B, float *C, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements)
  {
    C[i] = A[i] + B[i];
  }
}

// Stream for the thread's GPU Operations
hipStream_t stream;

// Memory regions
float *hA, *hB, *hC;
float *dA, *dB, *dC;
size_t vector_bytes;
int v_threadsPerBlock;
int v_blocksPerGrid;

void* Initialize(int sync_level) {
  /*
   * The sync_level parameter is an integer that indicates the desired level of
   * synchronization used by the GPU driver (values defined below).  The
   * specified level is used in hipSetDeviceFlags() to set the level
   * prior to initialization.
   */
  switch (sync_level) {
    case 0:
      hipSetDeviceFlags(hipDeviceScheduleSpin);
      break;
    case 1:
      hipSetDeviceFlags(hipDeviceScheduleYield);
      break;
    case 2:
      hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
      break;
    default:
      fprintf(stderr, "Unknown sync level: %d\n", sync_level);
      break;
  }
  hipSetDevice(0);
  hipStreamCreate(&stream);
  return NULL;
}

void MallocCPU(int numElements, void *thread_data) {
  vector_bytes = numElements * sizeof(float);

  // Host allocations in pinned memory
  // Allocate the host input vector A
  hipError_t err = hipHostMalloc((void **) &hA, vector_bytes);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate host vector A (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the host input vector B
  err = hipHostMalloc((void **) &hB, vector_bytes);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate host vector B (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the host output vector C
  err = hipHostMalloc((void **)&hC, vector_bytes);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate host vector C (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    hA[i] = rand()/(float)RAND_MAX;
    hB[i] = rand()/(float)RAND_MAX;
  }
  v_threadsPerBlock = 256;
  v_blocksPerGrid = (numElements + v_threadsPerBlock - 1) / v_threadsPerBlock;
}

void MallocGPU(int numElements, void *thread_data) {
  // Allocate the device input vector A
  hipError_t err = hipMalloc((void **)&dA, vector_bytes);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the device input vector B
  err = hipMalloc((void **)&dB, vector_bytes);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the device output vector C
  err = hipMalloc((void **)&dC, vector_bytes);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
    return;
  }
}

void CopyIn(int numElements, void *thread_data) {
  // copy the A and B vectors from Host to Device memory
  // these calls are asynchronous so only the lock of CE can be handled in the wrapper
  hipError_t err = hipMemcpyAsync(dA, hA, vector_bytes, hipMemcpyHostToDevice, stream);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  err = hipMemcpyAsync(dB, hB, vector_bytes, hipMemcpyHostToDevice, stream);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(stream);
}

void Exec(int numElements, void *thread_data) {
  hipError_t err = hipSuccess;

  // Launch the Vector Add CUDA Kernel
  // lock of EE is handled in wrapper for hipLaunchByPtr()
  vectorAdd<<<v_blocksPerGrid, v_threadsPerBlock, 0, stream>>>(dA, dB, dC, numElements);

  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
    return;
  }
  // synchronize with the stream after kernel execution
  // the wrapper for this function releases any lock held (EE here)
  hipStreamSynchronize(stream);
}

void CopyOut(void *thread_data) {
  // Copy the result vector from Device to Host memory
  // This call is asynchronous so only the lock of CE can be handled in the wrapper
  hipError_t err = hipMemcpyAsync(hC, dC, vector_bytes, hipMemcpyDeviceToHost, stream);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(stream);
}

void FreeGPU(void *thread_data) {
  // Free device global memory for inputs A and B and result C
  hipError_t err = hipFree(dA);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  err = hipFree(dB);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  err = hipFree(dC);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
    return;
  }
}

void FreeCPU(void *thread_data) {
  // Free host memory that was pinned
  hipHostFree(hA);
  hipHostFree(hB);
  hipHostFree(hC);
}

void Finish(void *thread_data) {
  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  hipError_t err = hipDeviceReset();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
  }
}
