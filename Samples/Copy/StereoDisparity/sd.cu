/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* A CUDA program that demonstrates how to compute a stereo disparity map using
 *   SIMD SAD (Sum of Absolute Difference) intrinsics
 */

/*
 * The program's performance is dominated by 
 * the computation on the execution engine (EE) while memory copies 
 * between Host and Device using the copy engine (CE) are significantly
 * less time consuming.
 *
 * This version uses a user allocated stream and asynchronous memory
 * copy operations (hipMemcpyAsync()).  Cuda kernel invocations on the
 * stream are also asynchronous.  hipStreamSynchronize() is used to 
 * synchronize with both the copy and kernel executions.  Host pinned
 * memory is not used because the copy operations are not a significant 
 * element of performance.
 *
 * The program depends on two input files containing the image 
 * representations for the left and right stereo images 
 * (stereo.im0.640x533.ppm and stereo.im1.640x533.ppm)
 * which must be in the directory with the executable.
 *
 */

#include <errno.h>
#include <math.h>
#include <sched.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/mman.h>
#include <sys/types.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
extern "C" {
#include "gpusync.h"
}
#include "sd_kernel.cuh"

// Relative path to images
static const char fname0[] = "../Samples/Copy/StereoDisparity/data/stereo.im0.640x533.ppm";
static const char fname1[] = "../Samples/Copy/StereoDisparity/data/stereo.im1.640x533.ppm";

// Holds per-thread state for this algorithm.
typedef struct {
  hipStream_t stream;
  // Host Memory
  unsigned int *h_odata;
  unsigned char *h_img0;
  unsigned char *h_img1;
  // Device memory
  unsigned int *d_odata;
  unsigned int *d_img0;
  unsigned int *d_img1;
  // Kernel execution parameters
  unsigned int w, h;
  dim3 numThreads;
  dim3 numBlocks;
  unsigned int numData;
  unsigned int memSize;
  hipTextureObject_t texture_right;
  hipTextureObject_t texture_left;
  // Search parameters
  int minDisp;
  int maxDisp;
} ThreadContext;

int iDivUp(int a, int b) {
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

// Override helper_image.h
inline bool loadPPM4ub(const char *file, unsigned char **data,
  unsigned int *w, unsigned int *h) {
  unsigned char *idata = 0;
  unsigned int channels;
  if (!__loadPPM(file, &idata, w, h, &channels)) {
    free(idata);
    return false;
  }
  // pad 4th component
  int size = *w * *h;
  // keep the original pointer
  unsigned char *idata_orig = idata;
  checkCudaErrors(hipHostMalloc(data, sizeof(unsigned char) * size * 4));
  unsigned char *ptr = *data;
  for (int i = 0; i < size; i++) {
    *ptr++ = *idata++;
    *ptr++ = *idata++;
    *ptr++ = *idata++;
    *ptr++ = 0;
  }
  free(idata_orig);
  return true;
}

void* Initialize(int sync_level) {
  ThreadContext *g;
  switch (sync_level) {
  case 0:
    hipSetDeviceFlags(hipDeviceScheduleSpin);
    break;
  case 1:
    hipSetDeviceFlags(hipDeviceScheduleYield);
    break;
  case 2:
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    break;
  default:
    printf("Unknown sync level: %d\n", sync_level);
    break;
  }
  hipSetDevice(0);
  hipFree(0);
  checkCudaErrors(hipHostMalloc(&g, sizeof(ThreadContext)));
  /*
  g = (ThreadContext *) malloc(sizeof(ThreadContext));
  if (!g) {
    printf("Failed allocating thread context.\n");
    exit(1);
  }
  */
  g->minDisp = -16;
  g->maxDisp = 0;
  hipStreamCreate(&(g->stream));
  return g;
}

void MallocCPU(int numElements, void *thread_data) {
  ThreadContext *g = (ThreadContext*) thread_data;
  // Load image data
  // functions allocate memory for the images on host side
  // initialize pointers to NULL to request lib call to allocate as needed
  // PPM images are loaded into 4 byte/pixel memory (RGBX)
  g->h_img0 = NULL;
  g->h_img1 = NULL;
  if (!loadPPM4ub(fname0, &(g->h_img0), &(g->w), &(g->h))) {
    fprintf(stderr, "Failed to load <%s>\n", fname0);
    exit(-1);
  }
  if (!loadPPM4ub(fname1, &(g->h_img1), &(g->w), &(g->h))) {
    fprintf(stderr, "Failed to load <%s>\n", fname1);
    exit(-1);
  }
  // set up parameters used in the rest of program
  g->numThreads = dim3(blockSize_x, blockSize_y, 1);
  g->numBlocks = dim3(iDivUp(g->w, g->numThreads.x), iDivUp(g->h,
    g->numThreads.y));
  g->numData = g->w * g->h;
  g->memSize = sizeof(int) * g->numData;

  // allocate memory for the result on host side
  checkCudaErrors(hipHostMalloc(&(g->h_odata), g->memSize));
}


void MallocGPU(int unused, void *thread_data) {
  ThreadContext *g = (ThreadContext*) thread_data;
  hipResourceDesc left_resource, right_resource;
  hipTextureDesc texture_desc;
  hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned int>();
  // allocate device memory for inputs and result
  checkCudaErrors(hipMalloc(&(g->d_odata), g->memSize));
  checkCudaErrors(hipMalloc(&(g->d_img0), g->memSize));
  checkCudaErrors(hipMalloc(&(g->d_img1), g->memSize));
  // Initialize texture objects.
  memset(&left_resource, 0, sizeof(left_resource));
  left_resource.resType = hipResourceTypePitch2D;
  left_resource.res.pitch2D.width = g->w;
  left_resource.res.pitch2D.height = g->h;
  left_resource.res.pitch2D.desc = desc;
  left_resource.res.pitch2D.pitchInBytes = g->w * 4;
  // The only difference between the left and right textures is the image
  memcpy(&right_resource, &left_resource, sizeof(left_resource));
  left_resource.res.pitch2D.devPtr = g->d_img0;
  right_resource.res.pitch2D.devPtr = g->d_img1;
  texture_desc.addressMode[0] = hipAddressModeClamp;
  texture_desc.addressMode[1] = hipAddressModeClamp;
  texture_desc.filterMode = hipFilterModePoint;
  texture_desc.readMode = hipReadModeElementType;
  checkCudaErrors(hipCreateTextureObject(&(g->texture_left), &left_resource,
    &texture_desc, NULL));
  checkCudaErrors(hipCreateTextureObject(&(g->texture_right), &right_resource,
    &texture_desc, NULL));
}

void CopyIn(int unused, void *thread_data) {
  ThreadContext *g = (ThreadContext*) thread_data;
  // copy host memory with images to device
  checkCudaErrors(hipMemcpyAsync(g->d_img0, g->h_img0, g->memSize,
    hipMemcpyHostToDevice, g->stream));
  checkCudaErrors(hipMemcpyAsync(g->d_img1, g->h_img1, g->memSize,
    hipMemcpyHostToDevice, g->stream));
  // copy host memory that was set to zero to initialize device output
  checkCudaErrors(hipMemcpyAsync(g->d_odata, g->h_odata, g->memSize,
    hipMemcpyHostToDevice, g->stream));
  hipStreamSynchronize(g->stream);
}

void Exec(int unused, void *thread_data) {
  ThreadContext *g = (ThreadContext*) thread_data;
  stereoDisparityKernel<<<g->numBlocks, g->numThreads, 0, g->stream>>>(
    g->d_img0, g->d_img1, g->d_odata, g->w, g->h, g->minDisp, g->maxDisp,
    g->texture_left, g->texture_right);
  hipStreamSynchronize(g->stream);
  getLastCudaError("Kernel execution failed");
}

void CopyOut(void *thread_data) {
  ThreadContext *g = (ThreadContext*) thread_data;
  checkCudaErrors(hipMemcpyAsync(g->h_odata, g->d_odata, g->memSize,
    hipMemcpyDeviceToHost, g->stream));
  hipStreamSynchronize(g->stream);
}

void FreeGPU(void *thread_data) {
  ThreadContext *g = (ThreadContext*) thread_data;
  checkCudaErrors(hipFree(g->d_odata));
  checkCudaErrors(hipFree(g->d_img0));
  checkCudaErrors(hipFree(g->d_img1));
}

void FreeCPU(void *thread_data) {
  ThreadContext *g = (ThreadContext*) thread_data;
  hipHostFree(g->h_odata);
  hipHostFree(g->h_img0);
  hipHostFree(g->h_img1);
}

void Finish(void *thread_data) {
  ThreadContext *g = (ThreadContext*) thread_data;
  hipStreamSynchronize(g->stream);
  hipDestroyTextureObject(g->texture_right);
  hipDestroyTextureObject(g->texture_left);
  hipStreamDestroy(g->stream);
  hipHostFree(g);
  checkCudaErrors(hipDeviceReset());
}
