#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/types.h>
#include <sys/mman.h>
#include <unistd.h>
#include <sched.h>
#include <errno.h>

extern "C" {
#include "../gpusync.h"
}

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numElements)
  {
    C[i] = A[i] + B[i];
  }
}

// Stream for the thread's GPU Operations
hipStream_t stream;

// Memory regions
float *hA, *hB, *hC;
float *dA, *dB, *dC;
size_t vector_bytes;
int v_threadsPerBlock;
int v_blocksPerGrid;

void* Initialize(int sync_level) {
  /*
   * The sync_level parameter is an integer that indicates the desired level of
   * synchronization used by the GPU driver (values defined below).  The
   * specified level is used in hipSetDeviceFlags() to set the level
   * prior to initialization.
   */
  switch (sync_level) {
    case 0:
      hipSetDeviceFlags(hipDeviceScheduleSpin);
      break;
    case 1:
      hipSetDeviceFlags(hipDeviceScheduleYield);
      break;
    case 2:
      hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
      break;
    default:
      fprintf(stderr, "Unknown sync level: %d\n", sync_level);
      break;
  }

  // Set up zero copy
  hipSetDeviceFlags(hipDeviceMapHost);
  hipSetDevice(0);
  hipStreamCreate(&stream);
  return NULL;
}

void MallocCPU(int numElements, void *thread_data) {
  vector_bytes = numElements * sizeof(float);

  // Host allocations in pinned memory
  // Allocate the host input vector A
  hipError_t err = hipHostAlloc((void **) &hA, vector_bytes, hipHostMallocMapped);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate host vector A (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the host input vector B
  err = hipHostAlloc((void **) &hB, vector_bytes, hipHostMallocMapped);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate host vector B (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the host output vector C
  err = hipHostAlloc((void **)&hC, vector_bytes, hipHostMallocMapped);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate host vector C (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    hA[i] = rand()/(float)RAND_MAX;
    hB[i] = rand()/(float)RAND_MAX;
  }
  v_threadsPerBlock = 256;
  v_blocksPerGrid = (numElements + v_threadsPerBlock - 1) / v_threadsPerBlock;
}

void MallocGPU(int numElements, void *thread_data) {
  // Allocate the device input vector A
  hipError_t err = hipHostGetDevicePointer((void **)&dA, (void *) hA, 0);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the device input vector B
  err = hipHostGetDevicePointer((void **)&dB, (void *) hB, 0);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // Allocate the device output vector C
  err = hipHostGetDevicePointer((void **)&dC, (void *) hC, 0);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
    return;
  }

  // synchronize with the stream
  // the wrapper for this function releases any lock held (CE here)
  hipStreamSynchronize(stream);
}

void CopyIn(int numElements, void *thread_data) {
}

void Exec(int numElements, void *thread_data) {
  hipError_t err = hipSuccess;

  // Launch the Vector Add CUDA Kernel
  // lock of EE is handled in wrapper for hipLaunchByPtr()
  vectorAdd<<<v_blocksPerGrid, v_threadsPerBlock, 0, stream>>>(dA, dB, dC, numElements);

  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
    return;
  }
  // synchronize with the stream after kernel execution
  // the wrapper for this function releases any lock held (EE here)
  hipStreamSynchronize(stream);
}

void CopyOut(void *thread_data) {
}

void FreeGPU(void *thread_data) {
}

void FreeCPU(void *thread_data) {
  hipHostFree(hA);
  hipHostFree(hB);
  hipHostFree(hC);
}

void Finish(void *thread_data) {
  // clean up the user allocated stream
  hipStreamSynchronize(stream);
  hipStreamDestroy(stream);

  // Reset the device and return
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application returns
  hipError_t err = hipDeviceReset();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
  }
}
